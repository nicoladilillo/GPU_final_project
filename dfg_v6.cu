#include "hip/hip_runtime.h"
/**
 * Thi simpler example just to see how to use simple combination
 * in a parralel way on GPU. The elaborated combination don't follow 
 * the lexograpical order.
 * 
 * Use shared memory
 */

 #include <stdio.h>
 #include "dfg.h"

// #define TESTING_OP_AND_NODE
// #define TESTING
// #define TESTING_SCHEDULING

 __device__ int Choose(int n, int k)
 {
    if (n < k)
        return 0;  // special case
    if (n == k)
        return 1;

    int delta, iMax;

    if (k < n-k) // ex: Choose(100,3)
    {
        delta = n-k;
        iMax = k;
    }
    else         // ex: Choose(100,97)
    {
        delta = k;
        iMax = n-k;
    }

    int ans = delta + 1;

    for (int i = 2; i <= iMax; ++i)
    {
        ans = (ans * (delta + i)) / i;
    }

    return ans;
 } // Choose()
 
 // diaplay combination with given index
 __global__ void combination(const uint8_t n, int r, const int tot_comb, int const shared_memory_size,
    int const shared_memory_size_offset, const operation_GPU_t *Operation_init, const node_GPU_t *node_init,
    const int operation_number, const int node_number, const uint8_t resources_number, const int area_limit,
    uint8_t *final_best_combination, uint8_t *final_best_repetition,
    int *final_best_time, int *final_area_calculated)
 {
    int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (idx < tot_comb) {
        extern  __shared__ unsigned char s[];

        uint8_t i, j, z, k;

        int k_comb = r;
        int best_time = 0x7fffffff;
        int area_calculated = 0x7fffffff;
        int area = 0;
        
        uint8_t index = 0;
        uint8_t flag;
        int time;

        // This variable can be shared between threads in the same block
        node_GPU_t *node;
        operation_GPU_t *Operation;

        node = (node_GPU_t *) &(s[0]);            
        Operation = (operation_GPU_t *) &(s[(int) (node_number*sizeof(node_GPU_t))]);

        // use only one instanze for all nodes and operation information
        if (threadIdx.x == 0)
        {
            // TO.DO: include also the index node and resources

            // Copy operations information
            for(i = 0; i < operation_number; i++) 
                Operation[i] = Operation_init[i];

            // Copy nodes information
            for(i = 0; i < node_number; i++)
                node[i] = node_init[i];
        } 
        __syncthreads();

        // offset between group of array thread
        int memory_trace = (int) (shared_memory_size_offset + idx*shared_memory_size);
        
        // lenght k_comb
        uint8_t *final = (uint8_t *) &(s[memory_trace]);
        memory_trace += (int) (k_comb*sizeof(uint8_t));
        // lenght k_comb
        int *all_aree = (int *) &(s[memory_trace]);
        memory_trace += (int) (k_comb*sizeof(int));

        // can speed up the overall process coping in the local register
        // lenght operation_number
        resource_t *resources = (resource_t *) &(s[memory_trace]);
        memory_trace += (int) (resources_number*sizeof(resource_t));
        // variable used for operation covered
        uint8_t *operation_covered = (uint8_t *) &(s[memory_trace]);
        memory_trace += (int) (operation_number*sizeof(uint8_t));
        
        // variable used from scheduling node
        uint8_t *state = (uint8_t *) &(s[memory_trace]);
        memory_trace += (int) (node_number*sizeof(uint8_t));
        uint8_t *remain_time = (uint8_t *) &(s[memory_trace]);
        memory_trace += (int) (node_number*sizeof(uint8_t));
        uint8_t *id_resource = (uint8_t *) &(s[memory_trace]);
        memory_trace += (int) (node_number*sizeof(uint8_t));
        uint8_t *dependecies_level_satisfy = (uint8_t *) &(s[memory_trace]);
        memory_trace += (int) (node_number*sizeof(uint8_t));

        // lenght K_comb
        uint8_t *repeat = (uint8_t *) &(s[memory_trace]);
        memory_trace += (int) (k_comb*sizeof(uint8_t));
        uint8_t *best_combination = (uint8_t *) &(s[memory_trace]);
        memory_trace += (int) (k_comb*sizeof(uint8_t));
        uint8_t *best_repetition = (uint8_t *) &(s[memory_trace]);
        memory_trace += (int) (k_comb*sizeof(uint8_t));        

        for(i = 0; i < operation_number; i++) 
            operation_covered[i] = 0;

        uint8_t a = n;
        int b = k_comb;
        int x = idx; // x is the "dual" of m

        // calculate the combination
        for (i = 0; i < k_comb; i++)
        {
            --a;
            while (Choose((int) a,b) > x)
                --a;
            x = x - Choose((int) a,b);
            final[i] = a;
            b = b-1;
        }
        
        #ifdef TESTING_OP_AND_NODE
        // synchronize the local threads writing to the local memory cache
        __syncthreads();

        // check the best time
        if(idx == tot_comb-1)
        {
            printf("\nNODE inside kernel\n\n");
            for(i = 0; i < node_number; i++)
            {
                printf("%d) Node: %d - Operation: %d - Dependency_level: %d" , node[i].id_node, node[i].id_node, Operation[node[i].index_operation].operation_id, node[i].dependecies_level);
                if (node[i].dependecies_level != 0) {
                    printf(" - Dependecies: ");
                    if (node[i].dep1_index != EMPTY_INDEX)
                        printf("%d ", node[node[i].dep1_index].id_node);
                    if (node[i].dep2_index != EMPTY_INDEX)
                        printf("%d ", node[node[i].dep2_index].id_node);
                }
                if (node[i].index_next_node_occurency > 0) 
                {
                    printf(" - Next node:   ");
                    for(j = 0; j < node[i].index_next_node_occurency; j++)
                        printf("%d ", node[node[i].index_next_node[j]].id_node);
                }
                printf("\n");
            }

            printf("\nRESOURCES inside kernel\n\n");
            for(i = 0; i < operation_number; i++)
            {
                printf("For %d the node are: ", Operation[i].operation_id);
                for(j = 0; j < Operation[i].index_next_node_occurency; j++)
                    printf("%d ", Operation[i].index_next_node[j]);
                printf("\n");
                printf("\tID Area Speed Occ\n");
                for(j = 0; j < Operation[i].res_occurency; j++)
                {
                    printf("%d)\t%2d %4d %4d %4d\n", j, Operation[i].res[j].id, Operation[i].res[j].area, Operation[i].res[j].speed, Operation[i].res[j].occurency);
                }
            }
            printf("\n");
        }
        #endif

        #ifdef TESTING
        for(i = 0; i < tot_comb ; i++)
        {   
            __syncthreads();
            if(idx == i)
            {
                printf("\t%d) ", i);
                for(j = 0; j < k_comb; j++) 
                    printf("%d  ", final[j]);
                printf(" -- Memory is %d, from %d to %d \n", memory_trace,
                    (int)( shared_memory_size_offset + idx*shared_memory_size),
                    (int)( shared_memory_size_offset + (idx+1)*shared_memory_size));
            }
        }
        __syncthreads();
        #endif

        // assign resources and check if resources used cover all operations
        k = 0;
        area = 0;
        for(z = 0; z < k_comb; z++)
        {
            for(i = 0; i < operation_number; i++)
            {
                for(j = 0; j < Operation[i].res_occurency; j++)
                {
                    if (Operation[i].res[j].id == final[z])
                    {
                        operation_covered[i] = 1;
                        resources[final[z]] = Operation[i].res[j];
                        resources[final[z]].index_operation = i;
                        resources[final[z]].occurency = 1;
                        repeat[k] = 1;
                        all_aree[k++] = resources[final[z]].area;
                        area += resources[final[z]].area;
                    }
                }
            }
        }

        // all others repeated combination will be bigger 
        if (area > area_limit)
            index = k_comb;
        else {
            // work with repetition, with a maximum of area_limit
            for(i = 0; i < operation_number; i++)
            {
                if (operation_covered[i] != 1)
                    index = k_comb;
            }
        }
   
        // start repeat combination
        while(index != k_comb)
        {            
            // set occurency for each resources
            for(i = 0; i < k_comb; i++)
                resources[final[i]].occurency = repeat[i];

            // Set intial node property
            for(i = 0; i < node_number; i++)
            {
                dependecies_level_satisfy[i] = node[i].dependecies_level;
                state[i] = Idle;
                remain_time[i] = 0;
            }

            #ifdef TESING_SCHEDULING
            if(idx == 7 && k_comb == 3) {
                printf("START SCHEDULING WITH: \n");
                for(i = 0; i < k_comb; i++)
                    printf("\t%d %d\n", final[i], repeat[i]);
                printf("\n");
            }
            #endif

            flag = 0;
            if (area <= area_limit)
                flag = 1;
            time = -1;
            uint8_t index_node;
            while (flag)
            {
                #ifdef TESING_SCHEDULING
                if(idx == 7 && k_comb == 3) {
                    printf("START time %d\n", time+1);
                    printf("See IDLE node\n");
                }
                #endif
                flag = 0;
                // check between all operation and find node that can be scheduled or that are in execution, 
                // in case you find nothing this means that all nodes hande been scheduled
                for(i = 0; i < k_comb; i++) 
                {
                    // Put some node from idle to executed state
                    if(resources[final[i]].occurency > 0)
                    {

                        // TO DO 3: improvo exit cycle
                        for(j = 0; j < Operation[resources[final[i]].index_operation].index_next_node_occurency; j++)
                        {
                            index_node = Operation[resources[final[i]].index_operation].index_next_node[j];
                            // Check if exist a node that has parents scheduled and is in Idle state
                            if(dependecies_level_satisfy[index_node] == 0 && state[index_node] == Idle)
                            {
                                flag = 1;
                                // Associate the resources to the node and decrease the occurency
                                remain_time[index_node] = resources[final[i]].speed;
                                id_resource[index_node] = final[i];
                                state[index_node] = Execution;                               
                                resources[final[i]].occurency--;
                                #ifdef TESING_SCHEDULING
                                if(idx == 7 && k_comb == 3) {
                                    printf("Scheduling node %d at time %d with resources %d (remainign %d) - will finish at %d\n", index_node, time+1, 
                                        id_resource[index_node], resources[final[i]].occurency, time + remain_time[index_node]);
                                }
                                #endif
                                if (resources[final[i]].occurency == 0)
                                    break;
                            }
                        }
                    }
                }
                
                #ifdef TESING_SCHEDULING
                if(idx == 7 && k_comb == 3) {
                    printf("See EXECUTE node\n");
                }
                #endif

                // Put some node from idle to executed state
                for(j = 0; j < node_number; j++)
                {
                    // Check if exist a node that has parents scheduled and is in Idle state
                    if(state[j] == Execution)
                    {
                        flag = 1;
                        if (remain_time[j] == 1) 
                        {
                            #ifdef TESING_SCHEDULING
                            if(idx == 7 && k_comb == 3) {
                                printf("END node %d (op %d -- state %d) at time %d with resources %d\n", j, node[j].index_operation, state[index], time+1, id_resource[j]);
                            }
                            #endif
                            // Node terminates to use the resource and all his dependencies have to be free
                            state[j] = Finish;
                            resources[id_resource[j]].occurency++;
                            for(z = 0; z < node[j].index_next_node_occurency; z++)
                                dependecies_level_satisfy[node[j].index_next_node[z]]--; 
                        } else {
                            remain_time[j]--;
                            #ifdef TESING_SCHEDULING
                            if(idx == 7 && k_comb == 3) {
                                printf("Node %d (op %d -- state %d) at time %d with resources %d\n", j, node[j].index_operation, state[index], time+1, id_resource[j]);
                            }
                            #endif
                        }
                    }
                }
                
                #ifdef TESING_SCHEDULING
                if(idx == 7 && k_comb == 3) {
                    printf("End time %d\n\n", time+1);
                }
                #endif

                time++;
            } // End scheduling

            // see if a better result has been achived
            if(time > -1 && ((time < best_time) || (time == best_time && area < area_calculated)))
            //if(time > -1 && time < best_time)
            {
                for(i = 0; i < k_comb; i++) 
                {
                    // TO_DO2: save them in variable and then copy nack in shared memory
                    best_combination[i] = final[i];
                    best_repetition[i]  = repeat[i];
                }
                area_calculated = area;
                best_time = time;
            }


            // Calculate the new repetition and the new area value 
            // go haed only if are is lesser than area_limit
            index = 0;
            int max_repetition = 4;
            do {
                while(index < k_comb && ((area + all_aree[index] > area_limit) || repeat[index] == max_repetition))
                {
                    for(i = 1; i < repeat[index]; i++)
                        area -= all_aree[index];
                    repeat[index] = 1;
                    index++;
                }
                
                if (index < k_comb)
                {
                    repeat[index]++;
                    area += all_aree[index];
                }

            } while (index != k_comb && area > area_limit);
            
        }// End repeat combination
        
        // TO_DO1: save result using temporaly register
        final_best_time[idx] = best_time;
        final_area_calculated[idx] = area_calculated;
        for(i = 0; i < k_comb; i++)
        {
            final_best_combination[idx*k_comb+i] = best_combination[i];
            final_best_repetition[idx*k_comb+i] = best_repetition[i];
        }

        #ifdef TESTING
        for(j = 0; j < tot_comb; j++)
        {   
            int area_app, speed_app;
            __syncthreads();
            if(idx == j)
            {
                if (best_time == 0x7fffffff)
                {
                    printf("idx: %d --> No combination for ", j);
                    for(i = 0; i < k_comb; i++)
                        printf("%d  ", final[i]);
                } else {
                    printf("idx: %d - Best time: %d - area: %d\n", j, best_time, area_calculated);
                    for(i = 0; i < k_comb; i++)
                    {
                        for(z = 0; z < operation_number; z++)
                        {
                            for (k = 0; k < Operation[z].res_occurency; k++){
                                if(Operation[z].res[k].id== final_best_combination[idx*k_comb+i])
                                {
                                    area_app = Operation[z].res[k].area;
                                    speed_app = Operation[z].res[k].speed;
                                }
                            }
                        }
                            
                        printf("\tid: %d - occurency: %d - area: %d - speed: %d\n ", final_best_combination[idx*k_comb+i], final_best_repetition[idx*k_comb+i], area_app, speed_app);
                    }
                }
                printf("\n");   
            }
        }
        #endif
        
        // check the best time
        if(idx == 0)
        {
            __syncthreads();
            for(i = 1; i < tot_comb; i++)
            {   
                // TO-DO 2: use shared memory
                if (best_time > -1 && (best_time > final_best_time[i]
                    || (best_time == final_best_time[i] && area_calculated > final_area_calculated[i])))
                {
                    final_best_time[0] = final_best_time[i];
                    best_time = final_best_time[i];
                    final_area_calculated[0] = final_area_calculated[i];
                    area_calculated = final_area_calculated[i];
                    for(j = 0; j < k_comb; j++) 
                    {
                        final_best_combination[j] = final_best_combination[i*k_comb+j];
                        final_best_repetition[j] = final_best_repetition[i*k_comb+j];
                    }
                }
            }
        }
    } // End check if rigth thread
 } // End combination()
  

 int main(int argc, char const *argv[])
 {
    int app;            // for read int
    uint8_t i, j, k;    // use like iterator

    if (argc != 4)
    {
        printf("Error in argument, expected 3 but was %d!\n", argc-1);
        return -1;
    }

    /** Read resources */

    FILE *fp = fopen(argv[2], "r");
    if (fp == NULL) 
    {
        printf("Error file name: %s doesn't exist!\n", argv[2]);
        return -2;
    }
    
    // initilize resources
    uint8_t operation_number;
    fscanf(fp, "%d", &app);
    operation_number = app;

    printf("START reading operations\n");

    operation_t *Operation;
    Operation = (operation_t *)malloc(sizeof(operation_t)*operation_number);

    uint8_t resource_number = 0;
    uint8_t len;
    for(i = 0; i < operation_number; i++)
    {   
        fscanf(fp, "%s", Operation[i].name);
        fscanf(fp, "%d\n", &app);
        len = app;
        Operation[i].res_occurency = len;
        // assign id to operation in a increase order
        Operation[i].operation_id  = i;
        Operation[i].covered = 0; 
        Operation[i].used    = 0; 
        Operation[i].max_index_next_node_occurency = 4; 
        Operation[i].index_next_node = (uint8_t *)malloc(sizeof(uint8_t)*4);
        Operation[i].index_next_node_occurency = 0;
        Operation[i].res = (resource_t *)malloc(sizeof(resource_t)*len);
        // Read how many resources are avaiable for executed this operation and
        // read all its property (speed and area)
        for(j = 0; j < len; j++)
        {
            // use app to avoid problem whit int scanf that use 32 bits
            fscanf(fp, "%d", &Operation[i].res[j].area);
            fscanf(fp, "%d", &app);
            Operation[i].res[j].speed = app;
            Operation[i].res[j].id    = resource_number++;
        }
    }

    /** Read node_t */

    fp = fopen(argv[1], "r");
    if (fp == NULL) 
    {
        printf("Error file name: %s doesn't exist!\n", argv[1]);
        return -2;
    }

    // initilize the node
    uint8_t node_number;
    fscanf(fp, "%d", &app);
    node_number = app;

    printf("START reading nodes\n");

    node_t *node;
    node = (node_t *)malloc(sizeof(node_t)*node_number);

    uint8_t operation_used = 0;
    resource_number = 0;

    char temp1[8];
    char temp2[8];
    for(i = 0; i < node_number; i++) 
    {
        fscanf(fp, "%s", temp1);
        fscanf(fp, "%s", temp2);
        printf("%d %s %s\n", i, temp1, temp2);
        strcpy(node[i].name, temp1);
        node[i].id_node = i;
        node[i].state = Idle;
        node[i].dep1_index = EMPTY_INDEX;
        node[i].dep2_index = EMPTY_INDEX;
        node[i].index_next_node_occurency = 0;
        node[i].max_index_next_node_occurency = 4;
        node[i].index_next_node = (uint8_t * )malloc(sizeof(uint8_t)*4);
        node[i].index_next_node_occurency = 0;
        node[i].dependecies_level         = 0;
        node[i].dependecies_level_satisfy = 0;
        for(j = 0; j < operation_number; j++)
        {
            if (strcmp(temp2, Operation[j].name) == 0)
            {
                if(Operation[j].used == 0)
                {
                    Operation[j].used = 1;
                    operation_used++;
                }
                node[i].index_operation = j;
                // Add index to list of node in the propr operation
                if(Operation[j].max_index_next_node_occurency == Operation[j].index_next_node_occurency) 
                {
                    #ifdef TESTING
                    printf("\tREALLOC from %d ... ", Operation[j].max_index_next_node_occurency);
                    #endif
                    Operation[j].max_index_next_node_occurency *= 2;
                    #ifdef TESTING
                    printf("to %d ... ", Operation[j].max_index_next_node_occurency);
                    #endif
                    Operation[j].index_next_node = (uint8_t *)realloc((uint8_t *)Operation[j].index_next_node, sizeof(uint8_t)*Operation[j].max_index_next_node_occurency);
                    #ifdef TESTING
                    printf("done\n");
                    #endif
                }
                Operation[j].index_next_node[Operation[j].index_next_node_occurency++] = i;
                break;
            }
        }
        if (j == operation_number)
        {
            printf("Node with operation that doesn't exist!\n");
            return -2;
        }
    }
    
    // inizialize edge
    uint8_t len_edge;
    fscanf(fp, "%d", &app);
    len_edge = app;

    printf("START reading edge\n");
    uint8_t v, u;
    for(i = 0; i < len_edge; i++) 
    {
        // Read source node
        fscanf(fp, "%s", temp1);
        // Read destination node
        fscanf(fp, "%s", temp2);
        // Check the index of two nodes
        for (j = 0; j < node_number; j++)
        {
            if (strcmp(node[j].name, temp1) == 0)
                u = j;
            else if (strcmp(node[j].name, temp2) == 0)
                v = j;
        }
        
        // Put as one of next node for the one read first
        if(node[u].max_index_next_node_occurency == Operation[u].index_next_node_occurency) 
        {
            node[u].max_index_next_node_occurency *= 2;
            node[u].index_next_node = (uint8_t *)realloc((uint8_t *)node[u].index_next_node, sizeof(uint8_t)*node[u].max_index_next_node_occurency);
        }
        node[u].index_next_node[node[u].index_next_node_occurency++] = v;

        // Put like next node for the one read in second place
        if (node[v].dep1_index == EMPTY_INDEX) 
            node[v].dep1_index = u;
        else
            node[v].dep2_index = u;
        node[v].dependecies_level++;
        node[v].dependecies_level_satisfy++;
        
        printf("Node %s(%s) va in nodo %s(%s)\n",  
            node[u].name, Operation[node[u].index_operation].name, 
            node[v].name, Operation[node[v].index_operation].name);
    }

    /** Print all read data to check the correct assimilation*/

    printf("\nNODE\n\n");
    for(i = 0; i < node_number; i++)
    {
        printf("%d) Node: %s(%d) - Operation: %s" , node[i].id_node, node[i].name, node[i].id_node, Operation[node[i].index_operation].name);
        if (node[i].dependecies_level != 0) {
            printf(" - Dependecies: ");
            if (node[i].dep1_index != EMPTY_INDEX)
                printf("%s ", node[node[i].dep1_index].name);
            if (node[i].dep2_index != EMPTY_INDEX)
                printf("%s ", node[node[i].dep2_index].name);
        }
        if (node[i].index_next_node_occurency > 0) 
        {
            printf(" - Next node:   ");
            for(j = 0; j < node[i].index_next_node_occurency; j++)
                printf("%s ", node[node[i].index_next_node[j]].name);
        }
        printf("\n");
    }

    
    printf("\nRESOURCES\n\n");
    for(i = 0; i < operation_number; i++)
    {
        printf("For %s (USED %d) the node are: ", Operation[i].name, Operation[i].used);
        for(j = 0; j < Operation[i].index_next_node_occurency; j++)
            printf("%s ", node[Operation[i].index_next_node[j]].name);
        printf("\n");
        printf("\tID Area Speed\n");
        for(j = 0; j < Operation[i].res_occurency; j++)
        {
            printf("%d)\t%2d %4d %4d\n", j, Operation[i].res[j].id, Operation[i].res[j].area, Operation[i].res[j].speed);
        }
    }
    printf("\n");

    // Copy variable to use for GPU purpose
    node_GPU_t *node_GPU = (node_GPU_t *)malloc(node_number*sizeof(node_GPU_t));
    for (i = 0; i < node_number; i++)
    {
        node_GPU[i].id_node           = node[i].id_node;
        node_GPU[i].dep1_index        = node[i].dep1_index;
        node_GPU[i].dep2_index        = node[i].dep2_index;
        node_GPU[i].dependecies_level = node[i].dependecies_level;
        node_GPU[i].index_operation   = node[i].index_operation;
        node_GPU[i].index_next_node_occurency = node[i].index_next_node_occurency;
        node_GPU[i].index_next_node = (uint8_t *)malloc(sizeof(uint8_t)*node[i].index_next_node_occurency);
        for (j = 0; j < node[i].index_next_node_occurency; j++)
            node_GPU[i].index_next_node[j] = node[i].index_next_node[j];
    }

    operation_t *New_Operation     = (operation_t *)malloc(operation_used*sizeof(operation_t));
    operation_GPU_t *Operation_GPU = (operation_GPU_t *)malloc(operation_used*sizeof(operation_GPU_t));
    for(i = 0, resource_number = 0, k = 0; i < operation_number && k < operation_used; i++)
    {   
        if(Operation[i].used == 1)
        {
            New_Operation[k] = Operation[i];
            New_Operation[k].operation_id = k;
            Operation_GPU[k].operation_id = k;
            // copy next node occurency
            Operation_GPU[k].index_next_node_occurency = Operation[i].index_next_node_occurency;
            Operation_GPU[k].index_next_node           = Operation[i].index_next_node;
            for(j = 0; j < Operation[i].index_next_node_occurency; j++){
                node[Operation[i].index_next_node[j]].index_operation     = k;
                node_GPU[Operation[i].index_next_node[j]].index_operation = k;
            }
            // copy resources occurency
            Operation_GPU[k].res_occurency = Operation[i].res_occurency;
            Operation_GPU[k].res           = Operation[i].res;
            //Operation_GPU[k].res = (resource_t *)malloc(sizeof(resource_t)*Operation[i].res_occurency);
            for (j = 0; j < Operation[i].res_occurency; j++)
            {
                // Operation_GPU[k].res[j] = Operation[i].res[j];
                // Change id to resources and index operation
                Operation_GPU[k].res[j].id = resource_number++;
                Operation_GPU[k].res[j].index_operation = k;
            }
            // Operation_GPU[k].index_next_node_occurency = Operation[i].index_next_node_occurency;
            // Operation_GPU[k].index_next_node = (uint8_t *)malloc(sizeof(uint8_t)*Operation[i].index_next_node_occurency);
            // for (j = 0; j < Operation[i].index_next_node_occurency; j++)
            // {
            //     // if(i == 0)
            //     //     printf("%s %d %d %s\n", Operation[i].name, Operation[i].index_next_node_occurency, j, node[Operation[i].index_next_node[j]].name);
            //     Operation_GPU[k].index_next_node[j] = Operation[i].index_next_node[j];
            // }
            k++;
        }
    }
    operation_number = operation_used;
    Operation = New_Operation;

    printf("\nNODE to GPU\n\n");
    for(i = 0; i < node_number; i++)
    {
        printf("%d) Node: %s(%d) - Operation: %s(%d)" , node_GPU[i].id_node, node[node_GPU[i].id_node].name, node_GPU[i].id_node, Operation[node_GPU[i].index_operation].name, node_GPU[i].index_operation);
        if (node[i].dependecies_level != 0) {
            printf(" - Dependecies: ");
            if (node[i].dep1_index != EMPTY_INDEX)
                printf("%s ", node[node_GPU[i].dep1_index].name);
            if (node[i].dep2_index != EMPTY_INDEX)
                printf("%s ", node[node_GPU[i].dep2_index].name);
        }
        if (node[i].index_next_node_occurency > 0) 
        {
            printf(" - Next node:   ");
            for(j = 0; j < node_GPU[i].index_next_node_occurency; j++)
                printf("%s ", node[node_GPU[i].index_next_node[j]].name);
        }
        printf("\n");
    }

    printf("\nRESOURCES to GPU\n\n");
    for(i = 0; i < operation_number; i++)
    {
        printf("For %s(%d) the node are: ", Operation[Operation_GPU[i].operation_id].name, Operation_GPU[i].operation_id);
        for(j = 0; j < Operation[i].index_next_node_occurency; j++)
            printf("%s ", node[Operation_GPU[i].index_next_node[j]].name);
        printf("\n");
        printf("\tID Area Speed\n");
        for(j = 0; j < Operation_GPU[i].res_occurency; j++)
        {
            printf("%d)\t%2d %4d %4d\n", j, Operation[i].res[j].id, Operation[i].res[j].area, Operation[i].res[j].speed);
        }
    }
    printf("\n");

    // variables used for GPU
    int final_best_time, *dev_final_best_time;
    int final_area_calculated, *dev_final_area_calculated;
    uint8_t *final_best_combination, *dev_final_best_combination;
    uint8_t *final_best_repetition, *dev_final_best_repetition;
    operation_GPU_t *dev_Operation;
    node_GPU_t *dev_node;

    uint8_t *dev_app;

    // Allocatr GPU memory
    hipMalloc(&dev_Operation, operation_number*sizeof(operation_GPU_t));
    hipMemcpy(dev_Operation, Operation_GPU, operation_number*sizeof(operation_GPU_t), hipMemcpyHostToDevice);
    // Allocate the right quantity for store the proper dimension of array in each structure
    for(i = 0; i < operation_number; i++)
    {
        // Copy resources
        hipMalloc(&dev_app, Operation_GPU[i].res_occurency*sizeof(resource_t));
        hipMemcpy(dev_app, Operation_GPU[i].res, Operation_GPU[i].res_occurency*sizeof(resource_t), hipMemcpyHostToDevice);
        hipMemcpy(&(dev_Operation[i].res), &dev_app, sizeof(uint8_t *), hipMemcpyHostToDevice);
        // Copy index nodes
        hipMalloc(&dev_app, Operation_GPU[i].index_next_node_occurency*sizeof(uint8_t));
        hipMemcpy(dev_app, Operation_GPU[i].index_next_node, Operation_GPU[i].index_next_node_occurency*sizeof(uint8_t), hipMemcpyHostToDevice);
        hipMemcpy(&(dev_Operation[i].index_next_node), &dev_app, sizeof(uint8_t *), hipMemcpyHostToDevice);
    }

    hipMalloc(&dev_node, node_number*sizeof(node_GPU_t));
    hipMemcpy(dev_node, node_GPU, node_number*sizeof(node_GPU_t), hipMemcpyHostToDevice);

    for(i = 0; i < node_number; i++)
    {
        // Copy next index nodes
        hipMalloc(&dev_app, node_GPU[i].index_next_node_occurency*sizeof(uint8_t));
        hipMemcpy(dev_app, node_GPU[i].index_next_node, node_GPU[i].index_next_node_occurency*sizeof(uint8_t), hipMemcpyHostToDevice);
        hipMemcpy(&(dev_node[i].index_next_node), &dev_app, sizeof(uint8_t *), hipMemcpyHostToDevice);
    }

    // store the value for comparison
    uint8_t *best_final = (uint8_t *)malloc(sizeof(uint8_t)*(resource_number+1));   
    uint8_t *best_final_repetition = (uint8_t *)malloc(sizeof(uint8_t)*resource_number);
    int best_time = 0x7fffffff;
    int area_calculated = 0x7fffffff;
    int area_limit = atoi(argv[3]);

    printf("Number of possible resource is %d\n", resource_number);
    printf("k min is %d and k max is %d\n\n", operation_used, resource_number);

    int shared_memory_size;
    int tot_shared_memory;
    int offset_shared_memory_size = int (operation_number*sizeof(operation_GPU_t) + 
                                        node_number*sizeof(node_GPU_t));

    // to store the execution time of code
    double time_spent = 0.0;
 
    clock_t begin = clock();
    // how big are the cutset, modify it iteratively
    for(k = operation_used; k <= resource_number; k++) {
        // calculate number of combinations
        int n_f = 1; // nominatore fattoriale
        for (i = resource_number; i > k; i--) n_f *= i;
        int d_f = 1; // denominatore fattoriale
        for (i = 1; i <= resource_number - k ; i++) d_f *= i;
        int tot_comb = n_f/d_f;

        // sum of all vector inside kernel
        shared_memory_size = (int) (k*sizeof(uint8_t)*4 +
                                k*sizeof(int) +
                                operation_number*sizeof(uint8_t) +
                                resource_number*sizeof(resource_t) +
                                node_number*sizeof(uint8_t)*4);
        
        tot_shared_memory = offset_shared_memory_size + (shared_memory_size*tot_comb);
        printf("Number of total combination witk k equal to %d are: %d\n\tShared memory Total is %d -- Piece of shared memory is %d -- Offset is %d\n",
            k, tot_comb, tot_shared_memory, shared_memory_size, offset_shared_memory_size);

        hipMalloc(&dev_final_best_time, tot_comb*sizeof(int));
        
        hipMalloc(&dev_final_area_calculated, tot_comb*sizeof(int));

        hipMalloc(&dev_final_best_combination, k*tot_comb*sizeof(uint8_t));
        final_best_combination = (uint8_t *)malloc(k*sizeof(uint8_t));

        hipMalloc(&dev_final_best_repetition, k*tot_comb*sizeof(uint8_t));
        final_best_repetition = (uint8_t *)malloc(k*sizeof(uint8_t));

        // call kernel
        combination<<<1, tot_comb, tot_shared_memory>>>(resource_number, k, tot_comb, shared_memory_size,
            offset_shared_memory_size, dev_Operation, dev_node, operation_number, node_number, resource_number, area_limit,
            dev_final_best_combination, dev_final_best_repetition, dev_final_best_time, dev_final_area_calculated);
        
        hipDeviceSynchronize();

        hipMemcpy(&final_best_time, dev_final_best_time, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&final_area_calculated, dev_final_area_calculated, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(final_best_combination, dev_final_best_combination, k*sizeof(uint8_t), hipMemcpyDeviceToHost);
        hipMemcpy(final_best_repetition, dev_final_best_repetition, k*sizeof(uint8_t), hipMemcpyDeviceToHost);

        #ifdef TESTING
        printf("Best Combination: ");
        for(i = 0; i < k; i++)
            printf(" %2d %2d,", final_best_combination[i], final_best_repetition[i]);
        printf(" - Time: %d - Area: %d\n", final_best_time, final_area_calculated);
        printf("\n");
        #endif

        if(final_best_time > -1 && ((final_best_time < best_time) 
            || (final_best_time == best_time && final_area_calculated < area_calculated)))
        {
            for(i = 0; i < k; i++)
            {
                best_final[i] = final_best_combination[i];
                best_final_repetition[i] = final_best_repetition[i];
            }
            best_final[i] = EMPTY_INDEX;
            best_time = final_best_time;
            area_calculated = final_area_calculated;
        }

        hipFree(dev_final_best_time);
        hipFree(dev_final_area_calculated);
        hipFree(dev_final_best_combination);
        hipFree(dev_final_best_repetition);
    } // END For k subset
    clock_t end = clock();

    // calculate elapsed time by finding difference (end - begin) and
    // dividing the difference by CLOCKS_PER_SEC to convert to seconds
    time_spent += (double)(end - begin) / CLOCKS_PER_SEC;
    
    /** Print the best solution obtained */
    fp = fopen("log_v6.log", "a");
    
    time_t rawtime;
    struct tm * timeinfo;

    time ( &rawtime );
    timeinfo = localtime ( &rawtime );

    fprintf (fp, "--------------------------------------------------\n");
    fprintf (fp, "Current local time and date: %s\n", asctime(timeinfo) );
    fprintf (fp, "DFG is %s\n", argv[1]);
    fprintf (fp, "Reasources are %s\n", argv[2]);
    fprintf(fp, "Area Limit is %d\n", area_limit);
    fprintf (fp, "--------------------------------------------------\n\n");


    fprintf(fp, "\nArea Limit is %d\n", area_limit);
    fprintf(stdout, "\nArea Limit is %d\n", area_limit);
    fprintf(fp, "Best solution has time %d:\n", best_time);
    fprintf(stdout, "Best solution has time %d:\n", best_time);
    for(i = 0; i < resource_number && best_final[i] != EMPTY_INDEX; i++) 
    {
        for(j = 0; j < operation_number; j++) 
        {
            for(k = 0; k < Operation[j].res_occurency; k++) 
            {
                if (best_final[i] == Operation[j].res[k].id)
                {
                    fprintf(stdout, "\tOPERATION: %4s - ID RESOURCE: %2d - SPEED: %2d - AREA: %2d - OCCURENCY: %2d\n", 
                        Operation[j].name, Operation[j].res[k].id, Operation[j].res[k].speed, Operation[j].res[k].area, best_final_repetition[i]);
                    fprintf(fp, "\tOPERATION: %4s - ID RESOURCE: %2d - SPEED: %2d - AREA: %2d - OCCURENCY: %2d\n", 
                        Operation[j].name, Operation[j].res[k].id, Operation[j].res[k].speed, Operation[j].res[k].area, best_final_repetition[i]);

                }
            }
        }
    }

    fprintf(stdout, "Final area is %d\n", area_calculated);
    fprintf(fp, "Final area is %d\n", area_calculated);
  
    printf("\nThe elapsed time is %f seconds\n", time_spent);
    fprintf(fp,"\nThe elapsed time is %f seconds\n\n", time_spent);

    hipFree(dev_node);
    hipFree(dev_Operation);

    hipDeviceReset();
    return 0;
 }
  